#include <iostream>
#include <chrono>
#include <cstdlib>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 16

// Kernel mnożący macierze z użyciem pamięci shared (tiled multiplication)
__global__ void matrixMultiplyTiled(const double* A, const double* B, double* C, int n) {
    __shared__ double ds_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ double ds_B[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    double Pvalue = 0.0;

    int numTiles = (n + TILE_WIDTH - 1) / TILE_WIDTH;

    for (int ph = 0; ph < numTiles; ++ph) {
        if (Row < n && ph * TILE_WIDTH + tx < n)
            ds_A[ty][tx] = A[Row * n + ph * TILE_WIDTH + tx];
        else
            ds_A[ty][tx] = 0.0;

        if (Col < n && ph * TILE_WIDTH + ty < n)
            ds_B[ty][tx] = B[(ph * TILE_WIDTH + ty) * n + Col];
        else
            ds_B[ty][tx] = 0.0;

        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; ++k)
            Pvalue += ds_A[ty][k] * ds_B[k][tx];

        __syncthreads();
    }

    if (Row < n && Col < n)
        C[Row * n + Col] = Pvalue;
}

// Funkcja wywołująca kernel mnożenia macierzy
void multiplyMatricesCUDA(const double* d_A, const double* d_B, double* d_C, int size) {
    dim3 blockSize(TILE_WIDTH, TILE_WIDTH);
    dim3 gridSize((size + TILE_WIDTH - 1) / TILE_WIDTH,
                  (size + TILE_WIDTH - 1) / TILE_WIDTH);

    matrixMultiplyTiled<<<gridSize, blockSize>>>(d_A, d_B, d_C, size);
    hipDeviceSynchronize();
}

// Kernel inicjalizujący macierz jednostkową na GPU
__global__ void initIdentityMatrix(double* matrix, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n * n) {
        int row = idx / n;
        int col = idx % n;
        matrix[idx] = (row == col) ? 1.0 : 0.0;
    }
}

// Funkcja potęgowania macierzy z użyciem CUDA
void matrixPowerCUDA(const double* h_base, double* h_result, int size, int power) {
    size_t bytes = size * size * sizeof(double);

    // Alokacja pamięci na GPU
    double *d_acc, *d_result, *d_temp;
    hipMalloc(&d_acc, bytes);
    hipMalloc(&d_result, bytes);
    hipMalloc(&d_temp, bytes);

    // Skopiuj bazową macierz na GPU
    hipMemcpy(d_acc, h_base, bytes, hipMemcpyHostToDevice);

    // Inicjalizacja macierzy wynikowej jako jednostkowa na GPU
    int totalThreads = size * size;
    int block = 256;
    int grid = (totalThreads + block - 1) / block;
    initIdentityMatrix<<<grid, block>>>(d_result, size);
    hipDeviceSynchronize();

    // Realizacja szybkiego potęgowania macierzy na GPU
    while (power > 0) {
        if (power & 1) {
            multiplyMatricesCUDA(d_result, d_acc, d_temp, size);
            std::swap(d_result, d_temp);
        }
        multiplyMatricesCUDA(d_acc, d_acc, d_temp, size);
        std::swap(d_acc, d_temp);
        power >>= 1;
    }

    // Skopiuj wynik z GPU na CPU
    hipMemcpy(h_result, d_result, bytes, hipMemcpyDeviceToHost);

    // Zwolnij pamięć
    hipFree(d_acc);
    hipFree(d_result);
    hipFree(d_temp);
}

int main(int argc, char* argv[]) {
    int size = 200;
    int power = 100; // do testów

    if (argc >= 3) {
        size = std::atoi(argv[1]);
        power = std::atoi(argv[2]);
    }

    srand(42);
    double* A = new double[size * size];
    for (int i = 0; i < size * size; ++i)
        A[i] = rand() % 10;

    double* result = new double[size * size];

    auto start = std::chrono::high_resolution_clock::now();
    matrixPowerCUDA(A, result, size, power);
    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double, std::milli> duration = end - start;
    std::cout << "Time (CUDA tiled): " << duration.count() << " ms\n";

    delete[] A;
    delete[] result;
    return 0;
}
